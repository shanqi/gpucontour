#include <stdio.h>
#include <unistd.h>
#include <fcntl.h>
#include <cutil.h>

#include "spec.h"

hipArray* cuda_parabola_pixels = NULL;
texture<float, 2, hipReadModeElementType> tex_parabola_pixels;

__constant__ float const_parabola_filters[MAX_FILTER_LENGTH*MAX_FILTER_LENGTH*MAX_FILTER_ORIENTATION];

//float* cuda_parabola_filters;
//texture<float, 1, hipReadModeElementType> tex_parabola_filters;

float* cuda_parabola_trace;

#include <parabola_kernel.cu>

static inline void cuda_parabola_allocate(int norients, int width, int height, int border)
{
    int border_width = width+2*border;
    int border_height = height+2*border;

    hipChannelFormatDesc ch;
    ch = hipCreateChannelDesc<float>();

    //CUDA_SAFE_CALL(hipMallocArray(&cuda_parabola_pixels, &ch, border_width, border_height*norients) );
    int tret = hipMallocArray(&cuda_parabola_pixels, &ch, border_width, border_height*norients);
    printf( "tret=%d vs %d, cuda_parabola_pixels=%p, border_width=%d, border_height=%d, norients=%d.\n", tret, hipSuccess, cuda_parabola_pixels, border_width, border_height, norients );
    tex_parabola_pixels.addressMode[0] = hipAddressModeClamp;
    tex_parabola_pixels.addressMode[1] = hipAddressModeClamp;
    tex_parabola_pixels.filterMode = hipFilterModePoint;
    tex_parabola_pixels.normalized = 0;

   /*  CUDA_SAFE_CALL( */
/*         hipBindTextureToArray(tex_parabola_pixels, cuda_parabola_pixels) ); */

    CUDA_SAFE_CALL(
      hipMalloc((void**)&cuda_parabola_trace, width*height*norients*sizeof(float)) );
}

static inline void cuda_parabola_free()
{
    //CUDA_SAFE_CALL(hipUnbindTexture(tex_parabola_pixels));
    CUDA_SAFE_CALL(hipFreeArray(cuda_parabola_pixels));
    CUDA_SAFE_CALL(hipFree(cuda_parabola_trace));
    //CUDA_SAFE_CALL(hipFree(cuda_parabola_filters));
}

#define DEBUG_copy_cuda_parabola_buffers

static inline void copy_cuda_parabola_buffers(int norients, int width, int height, int border, float *devPixels, int filter_radius, int filter_length, float* host_filters)
{
    //printf("copy_cuda_parabola_buffers: %d, %d, %d, %d, %d, %d\n", norients, width, height, border, filter_radius, filter_length);
    int border_width = width+2*border;
    int border_height = height+2*border;

    // copy pixels
    //printf("border_width=%d, border_height=%d, norients=%d\n", border_width, border_height, norients);
    //printf("cuda_parabola_pixels=%p, devPixels=%p, sizeof(int)=%d, sizeof(float)=%d\n", cuda_parabola_pixels, devPixels, sizeof(int), sizeof(float));
#ifdef DEBUG_copy_cuda_parabola_buffers
    hipDeviceProp_t prop;
    int device;
    hipGetDevice(&device);
    int tret = hipGetDeviceProperties(&prop, device);
    printf("tret=%d, device=%d, memPitch=%lu\n", tret, device, prop.memPitch);
#endif

    CUDA_SAFE_CALL(hipMemcpy2DToArray(cuda_parabola_pixels, 0, 0, devPixels, border_width*sizeof(int), border_width*sizeof(int), border_height*norients, hipMemcpyDeviceToDevice) );

    // copy const buffers (filters)
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(const_parabola_filters), host_filters, norients*filter_length*filter_length*sizeof(float)) );

    //hipChannelFormatDesc channelMax = hipCreateChannelDesc<float>();
    //size_t offset = 0;
    //hipBindTexture(&offset, &tex_parabola_filters, cuda_parabola_filters, &channelMax, filter_length*filter_length*norients* sizeof(float));
}

static inline void cuda_parabola_kernel(int norients, int width, int height, int border, int filter_radius, int filter_length, float* devResult, int cuePitchInFloats)
{
    hipError_t err;
    const int block_size = 16;
	
    CUDA_SAFE_CALL(hipBindTextureToArray(tex_parabola_pixels, cuda_parabola_pixels) );
	int blocks1 = width/block_size + (width%block_size == 0 ? 0 : 1);
	int blocks2 = height*norients/block_size + ((height*norients)%block_size == 0 ? 0 : 1);
	// int blocks1 = width/block_size + 1;
	// int blocks2 = height*norients/block_size + 1;

    dim3 grid(blocks1, blocks2, 1);
    dim3 threads(block_size, block_size, 1);
	
	fprintf(stdout, "grid = [%d,%d,%d]\n", blocks1, blocks2, 1);
	
    parabolaKernel<<<grid, threads>>>(cuda_parabola_trace, width, height, height*norients, border, height+2*border, filter_radius, filter_length, filter_length*filter_length);

    if (hipSuccess != (err = hipDeviceSynchronize()))
    {
        fprintf(stderr, "TB ERROR at %s:%d \"%s\"\n",
            __FILE__, __LINE__,  hipGetErrorString(err));
    }

    if (hipSuccess != (err = hipGetLastError()))
    {
        fprintf(stderr, "TB ERROR at %s:%d \"%s\"\n",
            __FILE__, __LINE__, hipGetErrorString(err));
    }

    int nPixels = width * height;
    for(int i = 0; i < norients; i++) {
      hipMemcpy(devResult + cuePitchInFloats * i, cuda_parabola_trace + nPixels * i, nPixels*sizeof(float), hipMemcpyDeviceToDevice);
    }
    CUDA_SAFE_CALL(hipUnbindTexture(tex_parabola_pixels));
    //CUDA_SAFE_CALL(hipUnbindTexture(tex_parabola_filters));
 
/*     CUDA_SAFE_CALL( */
/*         hipMemcpy(host_gradient, cuda_parabola_trace, width*height*norients*sizeof(float), hipMemcpyDeviceToHost) ); */

#if 0
    int i, j, k;

    printf("gpu\n");
    for (i=0; i<norients; i++)
    {
        printf("%d orientation\n", i+1);
        for (j=0; j<height; j++)
        {
            for (k=0; k<width; k++)
            {
                printf("%9.6f ", host_gradient[k+j*width+i*width*height]);
            }
            printf("\n");
        }
    }
#endif
}

void gpu_parabola_init(int norients, int width, int height, int border)
{
    cuda_parabola_allocate(norients, width, height, border);
}

void gpu_parabola_cleanup()
{
    cuda_parabola_free();
}

void gpu_parabola(int norients, int width, int height, int border, float* devPixels, int filter_radius, int filter_length, float* filters, float* devResult, int cuePitchInFloats)
{
    copy_cuda_parabola_buffers(norients, width, height, border, devPixels, filter_radius, filter_length, filters);
    cuda_parabola_kernel(norients, width, height, border, filter_radius, filter_length, devResult, cuePitchInFloats);

#if 0
    for (int o=0; o<norients; o++)
    {
        int fd;
        char file[1024];
        sprintf(file, "bcg_%d_%d.dat", radius, o+1);
        fd = open(file, O_CREAT|O_WRONLY, 0666);
        write(fd, &host_gradient[o*width*height], width*height*sizeof(float));
        close(fd);
    }
#endif

}
